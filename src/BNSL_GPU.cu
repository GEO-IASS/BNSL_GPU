#include "BNSL_GPU.cuh"

// �ڵ�ȡֵ��Χ
int * valuesRange;

// ������
int nodesNum = 0;

// ����ȡֵ
int * samplesValues;

// ��������
int samplesNum;

// ����㼯�ϵĸ���
int parentSetNum;

// �ֲ��÷�Hash��
double * dev_lsTable;

// ������
int* globalBestGraph;
int* globalBestOrder;
double globalBestScore;

void BNSL_init(){

	// ��ȡ�ڵ���Ϣ
	readNodeInfo(&nodesNum, &valuesRange);

	// ��ȡ��������
	readSamples(&samplesValues, &samplesNum, nodesNum);

	// ��ʼ��GPU
	CUDA_CHECK_RETURN(hipDeviceReset(), "hipDeviceReset failed.");
}

void BNSL_calLocalScore(){

	int i;
	parentSetNum = 0;
	for (i = 0; i <= CONSTRAINTS; i++) {
		parentSetNum = parentSetNum + C(i, nodesNum - 1);
	}

	int * dev_valuesRange;
	int * dev_samplesValues;

	// ��GPU�з����ڴ�ռ�
	CUDA_CHECK_RETURN(hipMalloc(&dev_lsTable, nodesNum * parentSetNum * sizeof(double)), "hipMalloc failed: dev_lsTable.");
	CUDA_CHECK_RETURN(hipMalloc(&dev_valuesRange, nodesNum * sizeof(int)), "hipMalloc failed: dev_valuesRange.");
	CUDA_CHECK_RETURN(hipMalloc(&dev_samplesValues, samplesNum * nodesNum * sizeof(int)), "hipMalloc failed: dev_samplesValues.");

	// �����ݿ�����GPU�ڴ���
	CUDA_CHECK_RETURN(hipMemcpy(dev_valuesRange, valuesRange, nodesNum * sizeof(int), hipMemcpyHostToDevice), "hipMemcpy failed: valuesRange -> dev_valuesRange");
	CUDA_CHECK_RETURN(hipMemcpy(dev_samplesValues, samplesValues, samplesNum * nodesNum * sizeof(int), hipMemcpyHostToDevice), "hipMemcpy failed: samplesValues -> dev_samplesValues");

	// ����GPU����
	int threadNum = 64;
	int total = parentSetNum * nodesNum;
	int blockNum = (total - 1) / threadNum + 1;
	calcAllPossibleLocalScore_kernel << <blockNum, threadNum >> >(dev_valuesRange, dev_samplesValues, dev_lsTable, samplesNum, nodesNum, parentSetNum);
	CUDA_CHECK_RETURN(hipGetLastError(), "calcAllPossibleLocalScore_kernel launch failed.");
	CUDA_CHECK_RETURN(hipDeviceSynchronize(), "calcAllPossibleLocalScore_kernel failed on running.");

	// �ͷ���GPU�з�����ڴ�ռ�
	CUDA_CHECK_RETURN(hipFree(dev_valuesRange), "hipFree failed: dev_valuesRange.");
	CUDA_CHECK_RETURN(hipFree(dev_samplesValues), "hipFree failed: dev_samplesValues.");

	// ���մ���������ݵ��ڴ�
	free(valuesRange);
	free(samplesValues);
}

void BNSL_start(){

	int i, j, iter;
	int parentSetNumInOrder = 0;
	for (i = 0; i < nodesNum; i++){
		for (j = 0; j <= CONSTRAINTS&&j < i + 1; j++){
			parentSetNumInOrder += C(j, i);
		}
	}

	// ÿ���²���63��order������1������order
	int ordersNum = 128;

	// ��������
	int iterNum = 0;

	// ��ʼ������������������
	srand((unsigned int)time(NULL));

	// �������
	int seed = 1234;

	// GPU�д洢�²�������������
	int * dev_newOrders;
	CUDA_CHECK_RETURN(hipMalloc(&dev_newOrders, ordersNum * nodesNum * sizeof(int)), "hipMalloc failed: dev_newOrders.");
	// CPU�д洢��������
	int * newOrder = (int *)malloc(nodesNum * sizeof(int));
	// ��ʼ����������
	randInitOrder(newOrder, nodesNum);

	// GPU�д洢������������ĸ��ڵ㼯�ϵĵ÷�
	double * dev_parentSetScore;
	CUDA_CHECK_RETURN(hipMalloc(&dev_parentSetScore, ordersNum * parentSetNumInOrder * sizeof(double)), "hipMalloc failed: dev_result.");

	// GPU�д洢ÿ����㸸��㼯�ϵ���ߵ÷�
	double * dev_maxLocalScore;
	CUDA_CHECK_RETURN(hipMalloc(&dev_maxLocalScore, ordersNum * nodesNum * sizeof(double)), "hipMalloc failed: dev_maxLocalScore.");

	// GPU�д洢ÿ����������ĵ÷�
	double * dev_ordersScore;
	CUDA_CHECK_RETURN(hipMalloc(&dev_ordersScore, ordersNum * sizeof(double)), "hipMalloc failed: dev_ordersScore.");
	// CPU�д洢ÿ����������ĵ÷�
	double * ordersScore = (double *)malloc(ordersNum * sizeof(double));

	// GPU�д洢I�ĸ��ʷֲ�
	double *dev_prob;
	CUDA_CHECK_RETURN(hipMalloc(&dev_prob, ordersNum * sizeof(double)), "hipMalloc failed: dev_prob.");
	// CPU��I�ĸ��ʷֲ�
	double *prob = (double *)malloc(ordersNum * sizeof(double));

	// GPU�д洢I������
	int *dev_samples;
	CUDA_CHECK_RETURN(hipMalloc(&dev_samples, ordersNum * sizeof(int)), "hipMalloc failed: dev_samples.");
	// CPU��I������
	int *samples = (int *)malloc(ordersNum * sizeof(int));

	// CPU�д洢ȫ�����ŵ���������
	globalBestOrder = (int *)malloc(nodesNum * sizeof(int));
	globalBestScore = -FLT_MAX;

	// GPU�д洢hiprand�����״̬
	hiprandState *dev_curandState;
	CUDA_CHECK_RETURN(hipMalloc(&dev_curandState, ordersNum * sizeof(hiprandState)), "hipMalloc failed: dev_curandState.");
	// ��ʼ��hiprand�����״̬
	curandSetup_kernel << < 1, ordersNum >> >(dev_curandState, seed);
	CUDA_CHECK_RETURN(hipGetLastError(), "curandSetup_kernel launch failed.");

	calcCDFInit(ordersNum);

	for (iter = 0; iter < iterNum; iter++){
		// ��������µ���������
		CUDA_CHECK_RETURN(hipMemcpy(dev_newOrders, newOrder, nodesNum * sizeof(int), hipMemcpyHostToDevice), "hipMemcpy failed: newOrder -> dev_newOrders.");
		generateOrders_kernel << <1, ordersNum, nodesNum * 4 >> >(dev_newOrders, dev_curandState, nodesNum);
		CUDA_CHECK_RETURN(hipGetLastError(), "generateOrders_kernel launch failed.");

		//calcGPUTimeStart("calcOnePairPerThread_kernel: ");
		int totalPairNum = ordersNum * parentSetNumInOrder;
		int threadDimX = 128;
		int blockDim = (totalPairNum - 1) / threadDimX + 1;
		int blockDimX = 1;
		int blockDimY = 1;
		if (blockDim < 65535){
			blockDimX = 1;
			blockDimY = blockDim;
		}
		else{
			blockDimX = (blockDim - 1) / 65535 + 1;
			blockDimY = 65535;
		}
		dim3 gridDim(blockDimX, blockDimY);
		calcOnePairPerThread_kernel << <gridDim, threadDimX >> >(dev_lsTable, dev_newOrders, dev_parentSetScore, nodesNum, parentSetNum, parentSetNumInOrder);
		CUDA_CHECK_RETURN(hipGetLastError(), "calcOnePairPerThread_kernel launch failed.");
		//calcGPUTimeEnd();

		// ����ÿ�����÷���ߵĸ���㼯��
		calcMaxParentSetScoreForEachNode_kernel << <nodesNum, ordersNum >> >(dev_parentSetScore, dev_maxLocalScore, parentSetNumInOrder, nodesNum);
		CUDA_CHECK_RETURN(hipGetLastError(), "calcMaxLocalScoreForEachNode_kernel launch failed.");

		// ����������������ĵ÷�
		calcAllOrdersScore_kernel << <1, ordersNum >> >(dev_maxLocalScore, dev_ordersScore, nodesNum);
		CUDA_CHECK_RETURN(hipGetLastError(), "calcAllOrdersScore_kernel launch failed.");
		CUDA_CHECK_RETURN(hipMemcpy(ordersScore, dev_ordersScore, ordersNum * sizeof(double), hipMemcpyDeviceToHost), "hipMemcpy failed: dev_ordersScore -> ordersScore.");

		int *newOrders = (int *)malloc(ordersNum * nodesNum * sizeof(int));
		CUDA_CHECK_RETURN(hipMemcpy(newOrders, dev_newOrders, ordersNum * nodesNum * sizeof(int), hipMemcpyDeviceToHost), "test");

		// ����������ĵ÷�ת��ΪI���ۻ����ʷֲ�
		int maxId = calcCDF(ordersScore, prob);

		// �����Ž�Ƚ�
		if (ordersScore[maxId] > globalBestScore){
			CUDA_CHECK_RETURN(hipMemcpy(globalBestOrder, dev_newOrders + maxId * nodesNum, nodesNum * sizeof(int), hipMemcpyDeviceToHost), "hipMemcpy failed: dev_newOrders -> globalBestOrder");
			globalBestScore = ordersScore[maxId];
		}

		// �Ը�������Iȡ��
		CUDA_CHECK_RETURN(hipMemcpy(dev_prob, prob, ordersNum * sizeof(double), hipMemcpyHostToDevice), "hipMemcpy failed: prob -> dev_prob.");
		sample_kernel << <1, ordersNum, ordersNum * 8 >> >(dev_prob, dev_samples, dev_curandState, ordersNum);
		CUDA_CHECK_RETURN(hipGetLastError(), "sample_kernel launch failed.");
		CUDA_CHECK_RETURN(hipMemcpy(samples, dev_samples, ordersNum * sizeof(int), hipMemcpyDeviceToHost), "hipMemcpy failed: dev_samples -> samples.");

		int r = rand() % ordersNum;
		CUDA_CHECK_RETURN(hipMemcpy(newOrder, dev_newOrders + samples[r] * nodesNum, nodesNum * sizeof(int), hipMemcpyDeviceToHost), "hipMemcpy failed: dev_newOrders -> newOrder");
	}

	CUDA_CHECK_RETURN(hipFree(dev_newOrders), "hipFree failed: dev_newOrders.");
	CUDA_CHECK_RETURN(hipFree(dev_parentSetScore), "hipFree failed: dev_parentSetScore.");
	CUDA_CHECK_RETURN(hipFree(dev_maxLocalScore), "hipFree failed: dev_maxLocalScore.");
	CUDA_CHECK_RETURN(hipFree(dev_ordersScore), "hipFree failed: dev_ordersScore.");
	CUDA_CHECK_RETURN(hipFree(dev_prob), "hipFree failed: dev_prob.");
	CUDA_CHECK_RETURN(hipFree(dev_samples), "hipFree failed: dev_samples.");
	CUDA_CHECK_RETURN(hipFree(dev_curandState), "hipFree failed: dev_curandState.");
	free(newOrder);
	free(ordersScore);
	free(prob);
	free(samples);
	calcCDFFinish();
}

void BNSL_printResult() {
	/*
	printf("Bayesian Network learned:\n");
	for (int i = 0; i < nodesNum; i++){
		for (int j = 0; j < nodesNum; j++){
			printf("%d ", globalBestGraph[i*nodesNum + j]);
		}
		printf("\n");
	}
	*/

	printf("Best Score: %f \n", globalBestScore);
	printf("Best Topology: ");
	for (int i = 0; i < nodesNum; i++){
		printf("%d ", globalBestOrder[i]);
	}
	printf("\n");
}

void BNSL_finish(){
	CUDA_CHECK_RETURN(hipFree(dev_lsTable), "hipFree failed: dev_lsTable.");
	free(globalBestOrder);
	free(globalBestGraph);
}