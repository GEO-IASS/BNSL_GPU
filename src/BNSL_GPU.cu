#include "hip/hip_runtime.h"
#include "BNSL_GPU.cuh"

int * valuesRange;

int nodesNum = 0;

int * samplesValues;

int samplesNum;

int allParentSetNumPerNode;

double * dev_lsTable;

int* globalBestGraph;
int* globalBestOrder;
double globalBestScore;

void BNSL_init() {
	readNodeInfo(&nodesNum, &valuesRange);
	readSamples(&samplesValues, &samplesNum, nodesNum);
}

void BNSL_calcLocalScore() {

	int i;
	allParentSetNumPerNode = 0;
	for (i = 0; i <= CONSTRAINTS; i++) {
		allParentSetNumPerNode = allParentSetNumPerNode + C(i, nodesNum - 1);
	}

	int * dev_valuesRange;
	int * dev_samplesValues;
	int * dev_N;

	// calculate max different values number for all pair of child and parent set
	int valuesMaxNum = calcValuesMaxNum(valuesRange, nodesNum);

	CUDA_CHECK_RETURN(
			hipMalloc(&dev_lsTable,
					nodesNum * allParentSetNumPerNode * sizeof(double)),
			"hipMalloc failed: dev_lsTable.");
	CUDA_CHECK_RETURN(hipMalloc(&dev_valuesRange, nodesNum * sizeof(int)),
			"hipMalloc failed: dev_valuesRange.");
	CUDA_CHECK_RETURN(
			hipMalloc(&dev_samplesValues, samplesNum * nodesNum * sizeof(int)),
			"hipMalloc failed: dev_samplesValues.");
	CUDA_CHECK_RETURN(
			hipMalloc(&dev_N,
					nodesNum * allParentSetNumPerNode * valuesMaxNum
							* sizeof(int)), "hipMalloc failed: dev_N.");

	CUDA_CHECK_RETURN(
			hipMemcpy(dev_valuesRange, valuesRange, nodesNum * sizeof(int),
					hipMemcpyHostToDevice),
			"hipMemcpy failed: valuesRange -> dev_valuesRange");
	CUDA_CHECK_RETURN(
			hipMemcpy(dev_samplesValues, samplesValues,
					samplesNum * nodesNum * sizeof(int),
					hipMemcpyHostToDevice),
			"hipMemcpy failed: samplesValues -> dev_samplesValues");
	CUDA_CHECK_RETURN(
			hipMemset(dev_N, 0,
					nodesNum * allParentSetNumPerNode * valuesMaxNum
							* sizeof(int)), "hipMemset failed: dev_N.");

	int threadNum = 64;
	int total = allParentSetNumPerNode * nodesNum;
	int blockNum = (total - 1) / threadNum + 1;
	calcAllLocalScore_kernel<<<blockNum, threadNum>>>(dev_valuesRange,
			dev_samplesValues, dev_N, dev_lsTable, samplesNum, nodesNum,
			allParentSetNumPerNode, valuesMaxNum);
	CUDA_CHECK_RETURN(hipGetLastError(),
			"calcAllPossibleLocalScore_kernel launch failed.");
	CUDA_CHECK_RETURN(hipDeviceSynchronize(),
			"calcAllPossibleLocalScore_kernel failed on running.");

	CUDA_CHECK_RETURN(hipFree(dev_valuesRange),
			"hipFree failed: dev_valuesRange.");
	CUDA_CHECK_RETURN(hipFree(dev_samplesValues),
			"hipFree failed: dev_samplesValues.");

	free(valuesRange);
	free(samplesValues);
}

void BNSL_start() {

	int i, j, iter;
	int parentSetNumInOrder = 0;
	for (i = 0; i < nodesNum; i++) {
		for (j = 0; j <= CONSTRAINTS && j < i + 1; j++) {
			parentSetNumInOrder += C(j, i);
		}
	}

	int ordersNum = 128;

	int iterNum = 1;

	srand((unsigned int) time(NULL));

	int seed = 1234;

	int * dev_newOrders;
	CUDA_CHECK_RETURN(
			hipMalloc(&dev_newOrders, ordersNum * nodesNum * sizeof(int)),
			"hipMalloc failed: dev_newOrders.");

	int * newOrder = (int *) malloc(nodesNum * sizeof(int));

	randInitOrder(newOrder, nodesNum);

	double * dev_parentSetScore;
	CUDA_CHECK_RETURN(
			hipMalloc(&dev_parentSetScore,
					ordersNum * parentSetNumInOrder * sizeof(double)),
			"hipMalloc failed: dev_result.");

	double * dev_maxLocalScore;
	CUDA_CHECK_RETURN(
			hipMalloc(&dev_maxLocalScore,
					ordersNum * nodesNum * sizeof(double)),
			"hipMalloc failed: dev_maxLocalScore.");

	double * dev_ordersScore;
	CUDA_CHECK_RETURN(hipMalloc(&dev_ordersScore, ordersNum * sizeof(double)),
			"hipMalloc failed: dev_ordersScore.");

	double * ordersScore = (double *) malloc(ordersNum * sizeof(double));

	double *dev_prob;
	CUDA_CHECK_RETURN(hipMalloc(&dev_prob, ordersNum * sizeof(double)),
			"hipMalloc failed: dev_prob.");

	double *prob = (double *) malloc(ordersNum * sizeof(double));

	int *dev_samples;
	CUDA_CHECK_RETURN(hipMalloc(&dev_samples, ordersNum * sizeof(int)),
			"hipMalloc failed: dev_samples.");

	int *samples = (int *) malloc(ordersNum * sizeof(int));

	globalBestOrder = (int *) malloc(nodesNum * sizeof(int));
	globalBestScore = -FLT_MAX;

	hiprandState *dev_curandState;
	CUDA_CHECK_RETURN(
			hipMalloc(&dev_curandState, ordersNum * sizeof(hiprandState)),
			"hipMalloc failed: dev_curandState.");

	curandSetup_kernel<<<1, ordersNum>>>(dev_curandState, seed);
	CUDA_CHECK_RETURN(hipGetLastError(), "curandSetup_kernel launch failed.");

	calcCDFInit(ordersNum);

	for (iter = 0; iter < iterNum; iter++) {
		CUDA_CHECK_RETURN(
				hipMemcpy(dev_newOrders, newOrder, nodesNum * sizeof(int),
						hipMemcpyHostToDevice),
				"hipMemcpy failed: newOrder -> dev_newOrders.");
		generateOrders_kernel<<<1, ordersNum, nodesNum * 4>>>(dev_newOrders,
				dev_curandState, nodesNum);
		CUDA_CHECK_RETURN(hipGetLastError(),
				"generateOrders_kernel launch failed.");

		//calcGPUTimeStart("calcOnePairPerThread_kernel: ");
		int totalPairNum = ordersNum * parentSetNumInOrder;
		int threadDimX = 128;
		int blockDim = (totalPairNum - 1) / threadDimX + 1;
		int blockDimX = 1;
		int blockDimY = 1;
		if (blockDim < 65535) {
			blockDimX = 1;
			blockDimY = blockDim;
		} else {
			blockDimX = (blockDim - 1) / 65535 + 1;
			blockDimY = 65535;
		}
		dim3 gridDim(blockDimX, blockDimY);
		calcOnePairPerThread_kernel<<<gridDim, threadDimX>>>(dev_lsTable,
				dev_newOrders, dev_parentSetScore, nodesNum,
				allParentSetNumPerNode, parentSetNumInOrder);
		CUDA_CHECK_RETURN(hipGetLastError(),
				"calcOnePairPerThread_kernel launch failed.");
		//calcGPUTimeEnd();

		calcMaxParentSetScoreForEachNode_kernel<<<nodesNum, ordersNum>>>(
				dev_parentSetScore, dev_maxLocalScore, parentSetNumInOrder,
				nodesNum);
		CUDA_CHECK_RETURN(hipGetLastError(),
				"calcMaxLocalScoreForEachNode_kernel launch failed.");

		calcAllOrdersScore_kernel<<<1, ordersNum>>>(dev_maxLocalScore,
				dev_ordersScore, nodesNum);
		CUDA_CHECK_RETURN(hipGetLastError(),
				"calcAllOrdersScore_kernel launch failed.");
		CUDA_CHECK_RETURN(
				hipMemcpy(ordersScore, dev_ordersScore,
						ordersNum * sizeof(double), hipMemcpyDeviceToHost),
				"hipMemcpy failed: dev_ordersScore -> ordersScore.");

		int *newOrders = (int *) malloc(ordersNum * nodesNum * sizeof(int));
		CUDA_CHECK_RETURN(
				hipMemcpy(newOrders, dev_newOrders,
						ordersNum * nodesNum * sizeof(int),
						hipMemcpyDeviceToHost), "test");

		int maxId = calcCDF(ordersScore, prob);

		if (ordersScore[maxId] > globalBestScore) {
			CUDA_CHECK_RETURN(
					hipMemcpy(globalBestOrder,
							dev_newOrders + maxId * nodesNum,
							nodesNum * sizeof(int), hipMemcpyDeviceToHost),
					"hipMemcpy failed: dev_newOrders -> globalBestOrder");
			globalBestScore = ordersScore[maxId];
		}

		CUDA_CHECK_RETURN(
				hipMemcpy(dev_prob, prob, ordersNum * sizeof(double),
						hipMemcpyHostToDevice),
				"hipMemcpy failed: prob -> dev_prob.");
		sample_kernel<<<1, ordersNum, ordersNum * 8>>>(dev_prob, dev_samples,
				dev_curandState, ordersNum);
		CUDA_CHECK_RETURN(hipGetLastError(), "sample_kernel launch failed.");
		CUDA_CHECK_RETURN(
				hipMemcpy(samples, dev_samples, ordersNum * sizeof(int),
						hipMemcpyDeviceToHost),
				"hipMemcpy failed: dev_samples -> samples.");

		int r = rand() % ordersNum;
		CUDA_CHECK_RETURN(
				hipMemcpy(newOrder, dev_newOrders + samples[r] * nodesNum,
						nodesNum * sizeof(int), hipMemcpyDeviceToHost),
				"hipMemcpy failed: dev_newOrders -> newOrder");
	}

	CUDA_CHECK_RETURN(hipFree(dev_newOrders),
			"hipFree failed: dev_newOrders.");
	CUDA_CHECK_RETURN(hipFree(dev_parentSetScore),
			"hipFree failed: dev_parentSetScore.");
	CUDA_CHECK_RETURN(hipFree(dev_maxLocalScore),
			"hipFree failed: dev_maxLocalScore.");
	CUDA_CHECK_RETURN(hipFree(dev_ordersScore),
			"hipFree failed: dev_ordersScore.");
	CUDA_CHECK_RETURN(hipFree(dev_prob), "hipFree failed: dev_prob.");
	CUDA_CHECK_RETURN(hipFree(dev_samples), "hipFree failed: dev_samples.");
	CUDA_CHECK_RETURN(hipFree(dev_curandState),
			"hipFree failed: dev_curandState.");
	free(newOrder);
	free(ordersScore);
	free(prob);
	free(samples);
	calcCDFFinish();
}

void BNSL_printResult() {
	/*
	 printf("Bayesian Network learned:\n");
	 for (int i = 0; i < nodesNum; i++){
	 for (int j = 0; j < nodesNum; j++){
	 printf("%d ", globalBestGraph[i*nodesNum + j]);
	 }
	 printf("\n");
	 }
	 */

	printf("Best Score: %f \n", globalBestScore);
	printf("Best Topology: ");
	for (int i = 0; i < nodesNum; i++) {
		printf("%d ", globalBestOrder[i]);
	}
	printf("\n");
}

void BNSL_finish() {
	CUDA_CHECK_RETURN(hipFree(dev_lsTable), "hipFree failed: dev_lsTable.");
	free(globalBestOrder);
	free(globalBestGraph);
}
